#include "hip/hip_runtime.h"
#define NTHREADS 32

__global__ void 
cunn_ClassNLLCriterion_updateOutput_kernel
(
float* output, float *input, float *target, int nframe, int ndim, int sizeAverage
) { 
  __shared__ float shInputs[NTHREADS];
  register int i;
  
  shInputs[threadIdx.x] = .0;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    shInputs[threadIdx.x] += input[i*ndim+(int)target[i]-1];
  }
  __syncthreads();
  
  if (threadIdx.x == 0) {
    *output = .0;
    for (i = 0; i < NTHREADS; ++i)
      *output += shInputs[i];
    if (sizeAverage)
      *output /= nframe;
  }
}

__global__ void 
cunn_ClassNLLCriterion_updateGradInput_kernel
(
float* gradInput, float *target, int nframe, int ndim, float grad
) { 
  register int i;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    gradInput[i*ndim+(int)target[i]-1] = grad;
  }
}

static int cunn_ClassNLLCriterion_updateOutput(lua_State *L) {
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(input);
  float *input_data = THCudaTensor_data(input);
  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(target);
  float *target_data = THCudaTensor_data(target);
  
  THCudaStorage *output = THCudaStorage_newWithSize(1);

  if (input->nDimension == 1) {
    float tid;
    hipMemcpy(&tid, target_data, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(output->data, input_data+(int)tid-1, sizeof(float), hipMemcpyDeviceToDevice);
  }
  else if(input->nDimension == 2) {
    dim3 blocks(1);
    dim3 threads(NTHREADS);     
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    cunn_ClassNLLCriterion_updateOutput_kernel<<<blocks,threads>>>
      (output->data, input_data, target_data, input->size[0], input->size[1], sizeAverage);
  }
  else
    THArgCheck(0, 2, "vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  lua_pushnumber(L, -THCudaStorage_get(output, 0));
  lua_setfield(L, 1, "output");

  THCudaStorage_free(output);
  THCudaTensor_free(target);
  THCudaTensor_free(input);
  
  return 1;
}

static int cunn_ClassNLLCriterion_updateGradInput(lua_State *L) {
  
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(input);
  // float *input_data = THCudaTensor_data(input);
  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(target);
  float *target_data = THCudaTensor_data(target);

  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  gradInput = THCudaTensor_newContiguous(gradInput);
  float *gradInput_data = THCudaTensor_data(gradInput);

  float grad = -1.0;
  if (input->nDimension == 1) {
    float tid;
    hipMemcpy(&tid, target_data, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradInput_data+(int)tid-1, &grad, sizeof(float), hipMemcpyHostToDevice);
  }
  else if(input->nDimension == 2) {
    int nframe = input->size[0];
    int ndim = input->size[1];
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    if (sizeAverage) grad /= nframe;
    dim3 blocks(1);
    dim3 threads(NTHREADS);
    cunn_ClassNLLCriterion_updateGradInput_kernel<<<blocks,threads>>>
      (gradInput_data, target_data, nframe, ndim, grad);
  }
  else
    THArgCheck(0, 2, "vector or matrix expected");  

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(gradInput);
  THCudaTensor_free(target);
  THCudaTensor_free(input);
  
  return 1;
}


static const struct luaL_Reg cunn_ClassNLLCriterion__ [] = {
  {"ClassNLLCriterion_updateOutput", cunn_ClassNLLCriterion_updateOutput},
  {"ClassNLLCriterion_updateGradInput", cunn_ClassNLLCriterion_updateGradInput},
  {NULL, NULL}
};

static void cunn_ClassNLLCriterion_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_ClassNLLCriterion__, "nn");
  lua_pop(L,1);
}
