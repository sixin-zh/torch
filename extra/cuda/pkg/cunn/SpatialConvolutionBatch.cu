#include "hip/hip_runtime.h"
// 2D convolution (4D inputs)
// TODO scale in accGradParameters

#include "SpatialConvolutionBatch/updateOutput.cu"
#include "SpatialConvolutionBatch/updateGradInput.cu"
#include "SpatialConvolutionBatch/accGradParameters.cu"

static int cunn_SpatialConvolutionBatch_updateOutput(lua_State *L) {

  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch) tensor expected");

  long batchSize = input->size[0];
  long nInputPlane = input->size[1];
  long nInputRows = input->size[2];
  long nInputCols = input->size[3];

  long nOutputPlane = weight->size[0];
  long nOutputRows = (nInputRows - kH) / dH + 1;
  long nOutputCols = (nInputCols - kW) / dW + 1;

  luaL_argcheck(L, nInputPlane == weight->size[1], 2, "number of input plane not consistent");
  luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

  THCudaTensor_resize4d(output, batchSize, nOutputPlane, nOutputRows, nOutputCols);

  // all the data must be contiguous
  luaL_argcheck(L, THCudaTensor_isContiguous(input), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(weight), 1, "weight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(output), 1, "output must be contiguous");

  // raw pointers 
  float *input_data = THCudaTensor_data(input);
  float *weight_data = THCudaTensor_data(weight);
  float *output_data = THCudaTensor_data(output);

  /* /\* add bias first *\/ */
  /* long k,p; */
  /* THCudaTensor *outputPlane = THCudaTensor_new(); */
  /* THCudaTensor *outputBatch = THCudaTensor_new(); */
  /* for(p=0; p<input->size[0]; p++) { */
  /*   THCudaTensor_select(outputBatch, output, 0, p); */
  /*   for(k=0; k<nOutputPlane; k++) { */
  /*     THCudaTensor_select(outputPlane, outputBatch, 0, k); */
  /*     THCudaTensor_fill(outputPlane, THCudaTensor_get1d(bias, k)); */
  /*   } */
  /* } */
  /* THCudaTensor_free(outputPlane); */
  /* THCudaTensor_free(outputBatch); */

  // convolution
  spatialConvB_updateOutput(
    input_data, weight_data, output_data,
    batchSize, nInputPlane, nInputRows, nInputCols,
    nOutputPlane, nOutputRows, nOutputCols,
    kH, kW, dH, dW
  );

  return 1;
}

static int cunn_SpatialConvolutionBatch_updateGradInput(lua_State *L) {

  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  
  long batchSize = input->size[0];
  long nInputPlane = input->size[1];
  long nInputRows = input->size[2];
  long nInputCols = input->size[3];

  long nOutputPlane = weight->size[0];
  long nOutputRows = (nInputRows - kH) / dH + 1;
  long nOutputCols = (nInputCols - kW) / dW + 1;

  luaL_argcheck(L, nInputPlane == weight->size[1], 2, "number of input plane not consistent");
  luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

  // resize gradInput
  THCudaTensor_resize4d(gradInput, batchSize, nInputPlane, nInputRows, nInputCols);
  
  // all the data must be contiguous
  luaL_argcheck(L, THCudaTensor_isContiguous(input), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(weight), 1, "weight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(gradOutput), 1, "gradOutput must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(gradInput), 1, "gradInput must be contiguous");

  // raw pointers 
  float *gradInput_data = THCudaTensor_data(gradInput);
  float *weight_data = THCudaTensor_data(weight);
  float *gradOutput_data = THCudaTensor_data(gradOutput);

  // convolutions
  spatialConvB_updateGradInput(
    gradOutput_data, weight_data, gradInput_data, 
    batchSize, nInputPlane, nInputRows, nInputCols,
    nOutputPlane, nOutputRows, nOutputCols,
    kH, kW, dH, dW
  );

  return 1;
}

__global__ void _compute_gradBias(float *gradBias, float *gradOutput, float scale,
                                 int output_n, int output_h, int output_w)
{
  // each block does a plane
  int k = blockIdx.x;
  float *gradOutput_k = gradOutput + (k + threadIdx.y*output_n)*output_h*output_w;

  // offsets
  int i_start = threadIdx.x;
  int i_end = output_w*output_h;
  int i_step = blockDim.x;

  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int nthreads = blockDim.x * blockDim.y;

  // sum output plane k into partial sum array
  __shared__ float sums[512];
  sums[tid] = 0;
  for (int i=i_start; i<i_end; i+=i_step) {
    sums[tid] += gradOutput_k[i];
  }
  __syncthreads();

  // reduce
  if (tid == 0) {
    for (int i=0; i<nthreads; i++)
      gradBias[k] += scale*sums[i];
  }
}

static int cunn_SpatialConvolutionBatch_accGradParameters(lua_State *L) {

  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  float scale = luaL_optnumber(L, 4, 1);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  
  long batchSize = input->size[0];
  long nInputPlane = input->size[1];
  long nInputRows = input->size[2];
  long nInputCols = input->size[3];

  long nOutputPlane = gradWeight->size[0];
  long nOutputRows = (nInputRows - kH) / dH + 1;
  long nOutputCols = (nInputCols - kW) / dW + 1;

  luaL_argcheck(L, nInputPlane == gradWeight->size[1], 2, "number of input plane not consistent");
  luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

  // all the data must be contiguous: 
  luaL_argcheck(L, THCudaTensor_isContiguous(input), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(gradOutput), 1, "gradOutput must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(gradWeight), 1, "gradWeight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(gradBias), 1, "gradBias must be contiguous");

  // raw pointers 
  float *input_data = THCudaTensor_data(input);
  float *gradOutput_data = THCudaTensor_data(gradOutput);
  float *gradWeight_data = THCudaTensor_data(gradWeight);
  float *gradBias_data = THCudaTensor_data(gradBias);

  /* gradient to bias */
  dim3 blocks(nOutputPlane);
  long sl;
  for (sl=0; sl<gradOutput->size[0]; sl+=16) {
    int cst = 16;
    if ((cst+sl) > gradOutput->size[0]) cst = gradOutput->size[0] - sl;
    dim3 threads(16, cst);
    _compute_gradBias <<<blocks, threads>>> (gradBias_data, gradOutput_data + sl*gradOutput->stride[0], scale,
					    nOutputPlane, nOutputRows, nOutputCols);
  }

  /* gradient to kernel */
  spatialConvB_accGradParameters(
    input_data, gradOutput_data, gradWeight_data,
    batchSize, nInputPlane, nInputRows, nInputCols,
    nOutputPlane, nOutputRows, nOutputCols,
    kH, kW, dH, dW
  );

  return 0;
}

static const struct luaL_Reg cunn_SpatialConvolutionBatch__ [] = {
  {"SpatialConvolutionBatch_updateOutput", cunn_SpatialConvolutionBatch_updateOutput},
  {"SpatialConvolutionBatch_updateGradInput", cunn_SpatialConvolutionBatch_updateGradInput},
  {"SpatialConvolutionBatch_accGradParameters", cunn_SpatialConvolutionBatch_accGradParameters},
  {NULL, NULL}
};

static void cunn_SpatialConvolutionBatch_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialConvolutionBatch__, "nn");
  lua_pop(L,1);
}
